#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand.h>
#include <omp.h>

#define CUDA_CALL(x) do { if((x)!=hipSuccess) { \
    printf("Error at %s Line %d: %s\n",__FILE__,__LINE__,hipGetErrorString(x));}} while(0)
#define CURAND_CALL(x) do { if((x)!=HIPRAND_STATUS_SUCCESS) { \
    printf("Error at %s:%d\n",__FILE__,__LINE__);}} while(0)

__global__ void montecarlo(float* pt1, float* pt2, int* result, int total_threads, int n) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;

  while (tid < n) {
    float sq = pt1[tid]*pt1[tid] + pt2[tid]*pt2[tid];
    if (sq < 1) {
      result[tid] = 1;
    }
    else {
      result[tid] = 0;
    }
    tid += total_threads;
  }
}

int main(int argc, char * argv[]) {
  //omp_set_num_threads(2);

  int num_darts = 1<<25; //
  int N = 1<<27;  // can't be more than 2^30 or memory errors
  int Nx = 4; // omp_get_num_threads()*2; // must be even, can be arbitrarily large
  int num_threads = 256;
  int num_blocks = 128;
  double total_percent = 0.0;
  double pi;

  float** rand_host;
  float* rand_dev1;
  float* rand_dev2;

  rand_host = (float**) malloc(Nx*sizeof(float*));
  for (int i = 0; i < Nx; i++) {
    rand_host[i] = (float*) malloc(N*sizeof(float));
  }

  CUDA_CALL(hipMalloc(&rand_dev1, N*sizeof(float)));
  CUDA_CALL(hipMalloc(&rand_dev2, N*sizeof(float)));


  printf("%d\n", N);

  hiprandGenerator_t gen;

  /* Create pseudo-random number generator */
  CURAND_CALL(hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT));
  CURAND_CALL(hiprandSetPseudoRandomGeneratorSeed(gen, 1234ULL));

  // generate floats, copy back
  for (int i = 0; i < Nx; i++) {
    CURAND_CALL(hiprandGenerateUniform(gen, rand_dev1, N));
    CUDA_CALL(hipMemcpy(rand_host[i], rand_dev1, N * sizeof(float),hipMemcpyDeviceToHost));
  }
  CURAND_CALL(hiprandDestroyGenerator(gen));

  hipDeviceSynchronize();

  // make results vectors; 1 == in circle, 0 == outside circle
  int** results_host;
  int* results_dev;

  results_host = (int**) malloc(Nx/2*sizeof(int*));
  for (int i = 0; i < Nx/2; i++) {
    results_host[i] = (int*) malloc(N*sizeof(int));
  }

  CUDA_CALL(hipMalloc(&results_dev, N*sizeof(int)));

  // make streams; one per kernel?
  int num_streams = N/num_darts;

  printf("num streams %d\n", num_streams);
  printf("making streams\n");
  hipStream_t streams[num_streams];
  for (int i = 0; i < num_streams; i++) {
    CUDA_CALL(hipStreamCreate(&streams[i]));
  }

  printf("starting compute\n");
  for (int n = 0; n < Nx; n +=2) {
    printf("n is %d\n", n);
    #pragma omp parallel for num_threads(2)
    for (int i = 0; i < num_streams; i++) {
      // do a bunch of async memcpys to device
      //printf("first memcpy\n");
      CUDA_CALL(hipMemcpyAsync(&rand_dev1[i*num_darts], &rand_host[n][i*num_darts],
        num_darts*sizeof(float), hipMemcpyHostToDevice, streams[i]));
      //printf("second memcpy\n");
      CUDA_CALL(hipMemcpyAsync(&rand_dev2[i*num_darts], &rand_host[n+1][i*num_darts],
        num_darts*sizeof(float), hipMemcpyHostToDevice, streams[i]));
    }
    #pragma omp parallel for num_threads(2)
    for (int i = 0; i < num_streams; i++) {
      // launch kernels on streams
      //printf("calling kernel\n");
      montecarlo<<<num_blocks, num_threads, 0, streams[i]>>>(&rand_dev1[i*num_darts],
        &rand_dev2[i*num_darts], &results_dev[i*num_darts], num_blocks*num_threads, num_darts);
    }
    #pragma omp parallel for num_threads(2)
    for (int i = 0; i < num_streams; i++) {
      // do a bunch of async memcpys from device
      //printf("copy back\n");
      CUDA_CALL(hipMemcpyAsync(&results_host[n/2][i*num_darts], &results_dev[i*num_darts],
        num_darts*sizeof(int), hipMemcpyDeviceToHost, streams[i]));
    }
  }

  hipDeviceSynchronize();

  // sum results

  //int num_inside[omp_get_num_threads()];
  int num_inside[4];
  #pragma omp parallel for num_threads(2)
  for (int n = 0; n < Nx/2; n++) {
    num_inside[n] = 0;
    for (int i = 0; i < N; i++) {
      #pragma omp atomic
      num_inside[n] += results_host[n][i];
    }
    printf("num darts in circle %d: %d\n", n, num_inside[n]);
  }

  double percent[Nx/2];
  #pragma omp parallel for reduction (+:total_percent) num_threads(2)
  for (int n = 0; n < Nx/2; n++) {
    percent[n] = num_inside[n] / (double)N;
    #pragma omp atomic
    total_percent += percent[n];
  }

  //total_percent /= Nx/2;
  #pragma omp single
  { pi = total_percent * 4; }

  printf("pi is %f\n", pi);


  // free things
  hipFree(rand_dev1);
  hipFree(rand_dev2);
  hipFree(results_dev);

  for (int i = 0; i < Nx; i++) {
    free(rand_host[i]);
  }
  free(rand_host);

  for (int i = 0; i < Nx/2; i++) {
    free(results_host[i]);
  }
  free(results_host);
}
